#include "hip/hip_runtime.h"
#include <utils.h>
#include <thrust/unique.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <cuda_ms_filter/cuda_ms_filter.h>
#include <hip/driver_types.h>

__device__ inline float _get_element(const hipTextureObject_t img, int row, int col, int width, int height, int chn,
                                     float default_value = -999999.0f) {
    if (row >= 0 && row < height && col >= 0 && col < width) {
        return tex2D<float>(img, col, row + height * chn);
    } else {
        return default_value;
    }
}

template<int blk_w, int blk_h, int ch, int dis_range = 5, int max_iter = 5>
__global__ void _ms_filter(hipTextureObject_t in_tex,
                           float *output,
                           int width,
                           int height,
                           int pitch,
                           float color_range) {
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;
    int blk_x_idx = blockIdx.x * blockDim.x;
    int blk_y_idx = blockIdx.y * blockDim.y;
    int j = blk_x_idx + threadCol;
    int i = blk_y_idx + threadRow;
    if (j >= width || i >= height)
        return;
    int ic = i;
    int jc = j;
    int ic_old, jc_old;
    float L_old, U_old, V_old;
    float L = tex2D<float>(in_tex, j, i + height * 0);
    float U = tex2D<float>(in_tex, j, i + height * 1);
    float V = tex2D<float>(in_tex, j, i + height * 2);
    L = L * 100 / 255;
    U = U - 128;
    V = V - 128;

    float shift = 5.0f;
    int i2_from = max(0, i - dis_range), i2to = min(height, i + dis_range + 1);
    int j2_from = max(0, j - dis_range), j2to = min(width, j + dis_range + 1);
    __shared__ float neighbor_pixels[ch][
            dis_range + blk_h + dis_range][dis_range + blk_w + dis_range];
#pragma unroll
    for (int r = threadRow - dis_range; r < dis_range + blk_h; r += blk_h) {
        for (int c = threadCol - dis_range; c < blk_w + dis_range; c += blk_w) {
            if (r < dis_range + blk_h && c < dis_range + blk_w) {
                for (int chn = 0; chn < ch; chn++) {
                    neighbor_pixels[chn][c + dis_range][r + dis_range] =
                            _get_element(in_tex, blk_y_idx + r, blk_x_idx + c, width, height, chn);
//                    printf("(%d, %d): %f\n", r, c,
//                           _get_element(in_tex, blk_y_idx + r, blk_x_idx + c, width, height, chn));
                }
            }
        }
    }
    __syncthreads();
    for (int iters = 0; shift > 3 && iters < max_iter; iters++) {
        ic_old = ic;
        jc_old = jc;
        L_old = L;
        U_old = U;
        V_old = V;

        float mi = 0.0f;
        float mj = 0.0f;
        float mL = 0.0f;
        float mU = 0.0f;
        float mV = 0.0f;
        int num = 0.0f;

#pragma unroll 20
        for (int i2 = i2_from; i2 < i2to; i2++) {
            for (int j2 = j2_from; j2 < j2to; j2++) {
                float L2 = neighbor_pixels[0][j2 - blk_x_idx + dis_range][i2 - blk_y_idx + dis_range],
                        U2 = neighbor_pixels[1][j2 - blk_x_idx + dis_range][i2 - blk_y_idx + dis_range],
                        V2 = neighbor_pixels[2][j2 - blk_x_idx + dis_range][i2 - blk_y_idx + dis_range];
//                if (i == 0 && j == 0) {
//                    printf("%f %f %f %d %d %f \n", L2, U2, V2, j2, i2, tex2D<float>(in_tex, 0, 0 + height * 2));
//                }
                L2 = L2 * 100 / 255;
                U2 = U2 - 128;
                V2 = V2 - 128;
                float dL = L2 - L;
                float dU = U2 - U;
                float dV = V2 - V;
                if (dL * dL + dU * dU + dV * dV <= color_range) {
                    mi += i2;
                    mj += j2;
                    mL += L2;
                    mU += U2;
                    mV += V2;
                    num++;
                }
            }
        }
        float num_ = 1.f / num;

        L = mL * num_;
        U = mU * num_;
        V = mV * num_;
        ic = lround(mi * num_ + 0.5);
        jc = lround(mj * num_ + 0.5);
        int di = ic - ic_old;
        int dj = jc - jc_old;
        float dL = L - L_old;
        float dU = U - U_old;
        float dV = V - V_old;

        shift = di * di + dj * dj + dL * dL + dU * dU + dV * dV;
    }

    L = L * 255 / 100;
    U = U + 128;
    V = V + 128;
    output[(i + height * 0) * pitch / sizeof(float) + j] = L;
    output[(i + height * 1) * pitch / sizeof(float) + j] = U;
    output[(i + height * 2) * pitch / sizeof(float) + j] = V;
}

namespace CuMeanShift {
    template<int blk_w, int blk_h, int ch>
    void CudaMsFilter<blk_w, blk_h, ch>::ms_filter_luv(float *input,
                                                       float *output,
                                                       int width,
                                                       int height,
                                                       int pitch,
                                                       int dis_range,
                                                       float color_range,
                                                       int max_iter) {
        dim3 block(blk_w, blk_h);
        dim3 grid(CEIL(width, blk_w), CEIL(height, blk_h));

        /// create texture object
        hipResourceDesc res_desc;
        memset(&res_desc, 0, sizeof(res_desc));
        res_desc.resType = hipResourceTypePitch2D;
        res_desc.res.pitch2D.devPtr = input;
        res_desc.res.pitch2D.width = width;
        res_desc.res.pitch2D.height = height * ch;
        res_desc.res.pitch2D.pitchInBytes = pitch;
        res_desc.res.pitch2D.desc.f = hipChannelFormatKindFloat;
        res_desc.res.pitch2D.desc.x = 32; // bits per channel

        hipTextureDesc tex_desc;
        memset(&tex_desc, 0, sizeof(tex_desc));
        tex_desc.readMode = hipReadModeElementType;
        tex_desc.addressMode[0] = hipAddressModeBorder;
        tex_desc.addressMode[1] = hipAddressModeBorder;
        tex_desc.filterMode = hipFilterModePoint;

        hipTextureObject_t in_tex = 0;
        hipCreateTextureObject(&in_tex, &res_desc, &tex_desc, NULL);
        _ms_filter<blk_w, blk_h, ch, 5, 5> << < grid, block >> >
                                                      (in_tex, output, width, height, pitch, color_range);
        hipDeviceSynchronize();
        hipDestroyTextureObject(in_tex);
    }
}