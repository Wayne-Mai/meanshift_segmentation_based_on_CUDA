#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <utils.h>
#include <type_traits>
#include <cuda_flooding/cuda_flooding.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#ifdef __JETBRAINS_IDE__
#include <cuda_fake_headers.h>
#include <hip/driver_types.h>
#endif

__global__ void _initLabels(int *labels, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int gid = row * width + col;

    if (row >= height || col >= width)
        return;

    labels[gid] = gid;
}

__global__ void _propagateLabels(int *prop_id, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int gid = row * width + col;

    // flood and propagate labels

    if (row >= height || col >= width)
        return;

    prop_id[gid] = prop_id[prop_id[gid]];
}

__global__ void _setLabels(int *labels, int *prop_id, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int gid = row * width + col;

    if (row >= height || col >= width)
        return;

    labels[gid] = labels[prop_id[gid]];
}

template<int blk_w = 32, int blk_h = 32,
         int ch = 1, int rad = 4,
         typename std::enable_if<((((rad + blk_h) * blk_w * (ch+1)) +
                                   ((rad + blk_w) * blk_h * (ch+1)) <= 12288)), int>::type = 0>
__global__ void _sharedFlooding(hipTextureObject_t in_tex,
                                int *labels,
                                int *prop_id,
                                int width,
                                int height,
                                float color_range) {
    __shared__ float left_pixels[ch][blk_h][rad + blk_w];
    __shared__ float up_pixels[ch][rad + blk_h][blk_w];
    int thd_row = threadIdx.y;
    int thd_col = threadIdx.x;
    int blk_row = blockIdx.y * blockDim.y;
    int blk_col = blockIdx.x * blockDim.x;
    int row = blk_row + thd_row;
    int col = blk_col + thd_col;
    int gid = row * width + col;

    if (row >= height || col >= width)
        return;

    // load data to share memory
    for (int r = thd_row; r < rad + blk_h; r += blk_h) {
        for (int chn = 0; chn < ch; chn++) {
            up_pixels[chn][r][thd_col] = tex2D<float>(in_tex, col, (blk_row + r - rad) + chn * height);
        }
    }
    __syncthreads();

    for (int chn = 0; chn < ch; chn++) {
        left_pixels[chn][thd_row][thd_col + rad] = up_pixels[chn][thd_row + rad][thd_col];
    }

    if (thd_col < rad) {
        for (int chn = 0; chn < ch; chn++) {
            left_pixels[chn][thd_row][thd_col] = tex2D<float>(in_tex, col - rad, row + chn * height);
        }
    }
    __syncthreads();


    // find neighbor pixel with the minimum delta LUV
    int offset = 0;
    int new_offset;
    float min_delta_luv = 999999.0f;

    for (int r = 0; r < rad; r++) {
        // find pixel with min(delta LUV) in the left direction
        if (col - rad + r > 0) {
            float delta_luv = 0.0f;
            for (int chn = 0; chn < ch; chn++) {
                float delta = left_pixels[chn][thd_row][thd_col + r] -
                              left_pixels[chn][thd_row][thd_col + rad];
                delta_luv += delta * delta;
            }
            new_offset = row * width + (col - rad + r);;
            if (delta_luv < min_delta_luv) {
                min_delta_luv = delta_luv;
                offset = new_offset;
            }
            else if (delta_luv == min_delta_luv && offset > new_offset) {
                offset = new_offset;
            }
        }

        // find pixel with min(delta LUV) in the up direction
        if (row - rad + r > 0) {
            float delta_luv = 0.0f;
            for (int chn = 0; chn < ch; chn++) {
                float delta = up_pixels[chn][thd_row + r][thd_col] -
                              up_pixels[chn][thd_row + rad][thd_col];
                delta_luv += delta * delta;
            }
            if (delta_luv < min_delta_luv) {
                min_delta_luv = delta_luv;
                offset = (row - rad + r) * width + col;
            }
            else if (delta_luv == min_delta_luv && offset > new_offset) {
                offset = new_offset;
            }
        }

    }

    if (min_delta_luv < color_range) {
        prop_id[gid] = offset;
    }
    else {
        prop_id[gid] = gid;
    }
}

namespace CuMeanShift {
    template<int blk_w, int blk_h, int ch, int rad>
    void CudaFlooding<blk_w, blk_h, ch, rad>::flooding(int *labels,
                                                       float *input,
                                                       int pitch,
                                                       int width,
                                                       int height,
                                                       int loops,
                                                       float color_range) {

        dim3 block(blk_w, blk_h);
        dim3 grid(CEIL(width, blk_w), CEIL(height, blk_h));

        /// create texture object
        hipResourceDesc res_desc;
        memset(&res_desc, 0, sizeof(res_desc));
        res_desc.resType = hipResourceTypePitch2D;
        res_desc.res.pitch2D.devPtr = input;
        res_desc.res.pitch2D.width = width;
        res_desc.res.pitch2D.height = height * ch;
        res_desc.res.pitch2D.pitchInBytes = pitch;
        res_desc.res.pitch2D.desc.f = hipChannelFormatKindFloat;
        res_desc.res.pitch2D.desc.x = 32; // bits per channel

        hipTextureDesc tex_desc;
        memset(&tex_desc, 0, sizeof(tex_desc));
        tex_desc.readMode = hipReadModeElementType;
        tex_desc.addressMode[0] = hipAddressModeBorder;
        tex_desc.addressMode[1] = hipAddressModeBorder;
        tex_desc.filterMode = hipFilterModePoint;

        hipTextureObject_t in_tex = 0;
        hipCreateTextureObject(&in_tex, &res_desc, &tex_desc, NULL);

        int *prop_id;
        hipMalloc(&prop_id, width * height * sizeof(int));
        _initLabels <<<grid, block>>> (labels, width, height);
        _sharedFlooding<blk_w, blk_h, ch, rad> <<< grid, block >>> (in_tex, labels, prop_id,
                width, height, color_range / loops);
        for (int i = 0; i < loops; i++) {
            _propagateLabels<<<grid, block>>> (prop_id, width, height);
        }
        _setLabels<<<grid, block>>> (labels, prop_id, width, height);
        hipDeviceSynchronize();
        hipDestroyTextureObject(in_tex);
        hipFree(prop_id);
    }
}

